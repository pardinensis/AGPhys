#include "hip/hip_runtime.h"
#include <cstdlib>
#include <hip/hip_runtime.h>
#include "hip/hip_vector_types.h"	// overload operators for floatN
#include "hip/hip_runtime_api.h"
#include "random.cuh"

typedef struct 
{
	float3 X;	// position
	float3 P;	// linear impulse
    float radius;
    float mass;
} ParticleState;

typedef struct
{
    float3 position;
    float3 velocity;
    float radius;
    float inv_mass;
} CollisionObject;

#define delta_spring 8000
#define delta_dashpot 30
#define delta_shear 10
#define delta_reflect 0.1

#define BOX_SIZE_X 10.0
#define BOX_SIZE_Z 10.0

#define SHARED_SIZE 1024

#define SAS


// integrators

//! euler1
//
//
__device__ ParticleState euler(ParticleState curState,
								float3 F,
								float w,
								float dT)
{
	float3 curV = w*curState.P;
	
	ParticleState result;
	result.X = curState.X + dT*curV;
	result.P = curState.P + dT*F;

	return result;
}

//! rungeKutta4XP
// - integrates an old position via rk4 regarding position and impulse
//
__device__ ParticleState rungeKutta4XP(ParticleState curState,
										float3 F,		// current linear force
										float w,		// inverse mass of particle
										float dT)		// step in time
{
	// time precalc
	float halfdt = dT * 0.5f;
	float sixthdt = dT / 6.0f;

	// cache
	float3 newV;

	// get current velocity
	float3 curV = w * curState.P; 
	
	// 1
	float3 A1_X_DT = curV;
	float3 A1_P_DT = F;
	newV = w * (curState.P + halfdt * A1_P_DT);
		
	//2
	float3 A2_X_DT = newV;
	float3 A2_P_DT = F;
	newV = w * (curState.P + halfdt * A2_P_DT);
	
	//3
	float3 A3_X_DT = newV;
	float3 A3_P_DT = F;
	newV = w * (curState.P + dT * A3_P_DT);
	
	// 4
	float3 A4_X_DT = newV;
	float3 A4_P_DT = F;
	
	// final update
	ParticleState result;
	result.X = curState.X + sixthdt * (A1_X_DT + 2.0f * (A2_X_DT + A3_X_DT) + A4_X_DT);
	result.P = curState.P + sixthdt * (A1_P_DT + 2.0f * (A2_P_DT + A3_P_DT) + A4_P_DT);
		
	return result;
}

__device__ float3 springDashpotForce(CollisionObject a, CollisionObject b) {
    float3 ab = b.position - a.position;
    float len_ab = length(ab);
    float3 n_ab = ab / len_ab;
    float3 v_rel = b.velocity - a.velocity;

    float3 f_spring = delta_spring * (a.radius + b.radius - len_ab) * n_ab;
    float3 f_dashpot = -delta_dashpot * v_rel;
    float3 f_shear = delta_shear * (v_rel - dot(v_rel, -n_ab) * (-n_ab));
    float3 f_total = f_spring + f_dashpot + f_shear;

    return f_total;
}

__device__ void respondSphereCollisions(float* ptVbo, float* f_tmp, int i, int j) {
    CollisionObject a;
    a.position = make_float3(ptVbo[12*i+0], ptVbo[12*i+1], ptVbo[12*i+2]);
    a.radius = ptVbo[12*i+9];
    a.inv_mass = 1.0f / ptVbo[12*i+11];
    a.velocity = make_float3(ptVbo[12*i+3], ptVbo[12*i+4], ptVbo[12*i+5]) * a.inv_mass;

    CollisionObject b;
    b.position = make_float3(ptVbo[12*j+0], ptVbo[12*j+1], ptVbo[12*j+2]);
    b.radius = ptVbo[12*j+9];
    b.inv_mass = 1.0f / ptVbo[12*j+11];
    b.velocity = make_float3(ptVbo[12*j+3], ptVbo[12*j+4], ptVbo[12*j+5]) * b.inv_mass;

    float3 f = springDashpotForce(a, b);

    atomicAdd(&(f_tmp[3*i+0]), -f.x);
    atomicAdd(&(f_tmp[3*i+1]), -f.y);
    atomicAdd(&(f_tmp[3*i+2]), -f.z);
    atomicAdd(&(f_tmp[3*j+0]), f.x);
    atomicAdd(&(f_tmp[3*j+1]), f.y);
    atomicAdd(&(f_tmp[3*j+2]), f.z);
}

__device__ void detectSphereCollisions(float *ptVbo, float* f_tmp, int numParticles, int i) {
    for (int j = i + 1; j < numParticles; ++j) {
        float3 Xi = make_float3(ptVbo[12*i+0], ptVbo[12*i+1], ptVbo[12*i+2]);
        float3 Xj = make_float3(ptVbo[12*j+0], ptVbo[12*j+1], ptVbo[12*j+2]);
        float ri = ptVbo[12*i+9];
        float rj = ptVbo[12*j+9];
        float3 toJ = Xj - Xi;
        float dist2 = toJ.x * toJ.x + toJ.y * toJ.y + toJ.z * toJ.z;
        if (dist2 < (rj + ri) * (rj + ri)) {
            respondSphereCollisions(ptVbo, f_tmp, i, j);
        }
    }
}

__device__ void detectPlaneCollisions(float *ptVbo, float* f_tmp, int i) {
    float3 position = make_float3(ptVbo[12*i+0], ptVbo[12*i+1], ptVbo[12*i+2]);
    float3 impulse = make_float3(ptVbo[12*i+3], ptVbo[12*i+4], ptVbo[12*i+5]);
    float radius = ptVbo[12*i+9];

    float epsilon = 0.00001;
    if (position.y - radius < 0 && impulse.y < 0) {
        ptVbo[12*i+1] = radius + epsilon;
        ptVbo[12*i+4] *= -delta_reflect;
    }
    if (position.x - radius < -BOX_SIZE_X/2 && impulse.x < 0) {
        ptVbo[12*i+0] = -BOX_SIZE_X/2 + radius + epsilon;
        ptVbo[12*i+3] *= -delta_reflect;
    }
    if (position.x + radius > BOX_SIZE_X/2 && impulse.x > 0) {
        ptVbo[12*i+0] = BOX_SIZE_X/2 - radius - epsilon;
        ptVbo[12*i+3] *= -delta_reflect;
    }
    if (position.z - radius < -BOX_SIZE_Z/2 && impulse.z < 0) {
        ptVbo[12*i+2] = -BOX_SIZE_Z/2 + radius + epsilon;
        ptVbo[12*i+5] *= -delta_reflect;
    }
    if (position.z + radius > BOX_SIZE_Z/2 && impulse.z > 0) {
        ptVbo[12*i+2] = BOX_SIZE_Z/2 - radius - epsilon;
        ptVbo[12*i+5] *= -delta_reflect;
    }
}

__device__ unsigned int id() {
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    seed = i;
    return i;
}


__global__ void box_collisions(float *ptVbo, float *f_tmp, int numParticles)
{
    unsigned int i = id();
    if(i >= numParticles) return;

    detectPlaneCollisions(ptVbo, f_tmp, i);
}

__global__ void sphere_collisions(float* ptVbo, float* f_tmp, int numParticles) {
    unsigned int i = id();
    if(i >= numParticles) return;

    detectSphereCollisions(ptVbo, f_tmp, numParticles, i);
}

__global__ void integrate(float* ptVbo, float* f_tmp, int numParticles, float dT) {
    unsigned int i = id();
    if(i >= numParticles) return;

    // some subtle random velocity changes
    // currentState.P += 0.0000001 * noise3D();

    // init current state
    ParticleState currentState;
    currentState.X.x = ptVbo[12*i + 0];
    currentState.X.y = ptVbo[12*i + 1];
    currentState.X.z = ptVbo[12*i + 2];
    currentState.P.x = ptVbo[12*i + 3];
    currentState.P.y = ptVbo[12*i + 4];
    currentState.P.z = ptVbo[12*i + 5];
    currentState.radius = ptVbo[12*i + 9];
    currentState.mass = ptVbo[12*i + 11];

    // retrieve collision changes
    float3 F = make_float3(f_tmp[3*i + 0], f_tmp[3*i + 1], f_tmp[3*i + 2]);

    // add some gravity
    F += make_float3(0, -10 * currentState.mass, 0);

    // call the intergrator
    ParticleState newState = rungeKutta4XP(currentState, F, 1.f/currentState.mass, dT);

    // save the changes to the buffer
    ptVbo[12*i + 0] = newState.X.x;
    ptVbo[12*i + 1] = newState.X.y;
    ptVbo[12*i + 2] = newState.X.z;
    ptVbo[12*i + 3] = newState.P.x;
    ptVbo[12*i + 4] = newState.P.y;
    ptVbo[12*i + 5] = newState.P.z;
}

/*
B 2 -- B 4 -- R 2 -- B 8 -- R 4 -- R 2 -- B 16 -- R 8 -- R 4 -- R 2

0/1 -- 0/3 -- 0/1 -- 0/7 -- 0/2 -- 0/1 -- 0/15 -- 0/4 -- 0/2 -- 0/1
2/3 -- 2/1 -- 2/3 -- 2/5 -- 3/1 -- 2/3 -- 2/13 -- 2/6 -- 3/1 -- 2/3
4/5 -- 4/7 -- 4/5 -- 4/3 -- 4/6 -- 4/5 -- 4/11 -- 5/1 -- 4/6 -- 4/5
6/7 -- 6/5 -- 6/7 -- 6/1 -- 7/5 -- 6/7 -- 6/09 -- 7/3 -- 7/5 -- 6/7

[B]
        x*b-1-2i
        x = int(2*i/b)*2+1
[R]
        back = (int(4i/r))%2
        if (!back)
            2i / 2i+(r/2)
        else
            2i+1 / 2i+1-(r/2)
*/

__global__ void reset_force_buffer(float* f_tmp, int numParticles) {
    int i = id();
    if(i >= numParticles) return;

    // reset force buffer
    f_tmp[3*i + 0] = 0;
    f_tmp[3*i + 1] = 0;
    f_tmp[3*i + 2] = 0;
}

__global__ void SAS_init(float *ptVbo, float* key, float* value, int numParticles) {
    int i = id();
    if(i >= numParticles) return;

    key[2*i+0] = ptVbo[12*i + 0] - ptVbo[12*i + 9];
    key[2*i+1] = ptVbo[12*i + 0] + ptVbo[12*i + 9];
    value[2*i+0] = i+1;
    value[2*i+1] = -(i+1);
}

__device__ void SAS_cas(float* k, float* v, unsigned int i1, unsigned int i2) {
    if ((i1 < i2) != (k[i1] < k[i2])) {
        float tmp = k[i1];
        k[i1] = k[i2];
        k[i2] = tmp;
        tmp = v[i1];
        v[i1] = v[i2];
        v[i2] = tmp;
    }
}

__global__ void SAS_sort_brown(float* k, float* v, unsigned int N, unsigned int B) {
    unsigned int i = id();
    if (2*i >= N) return;
    SAS_cas(k, v, 2*i, ((2*i/B)*2+1)*B-1-2*i);
}

__global__ void SAS_sort_red(float* k, float* v, unsigned int N, unsigned int R) {
    unsigned int i = id();
    if (2*i >= N) return;
    if ((4*i/R) % 2 == 0) {
        SAS_cas(k, v, 2*i, 2*i+R/2);
    }
    else {
        SAS_cas(k, v, 2*i+1, 2*i+1-R/2);
    }
}

//__device__ __forceinline__ bool about_equal(float a, float b) {
//    float epsilon = 0.000001;
//    return (a - b < epsilon) && (b - a < epsilon);
//}

__global__ void SAS_sphere_collisions(float* ptVbo, float* f_tmp, float* k, float* v, int numParticles) {
    unsigned int i = id();
    if (i >= 2*numParticles) return;

    if (v[i] < 0) return;

    unsigned int sphere1_id = v[i] - 1;
    for (unsigned int j = i+1;; ++j) {
        if (v[j] == -v[i]) break;
        if (v[j] < 0) continue;
        unsigned int sphere2_id = v[j] - 1;

        float3 Xi = make_float3(ptVbo[12*sphere1_id+0], ptVbo[12*sphere1_id+1], ptVbo[12*sphere1_id+2]);
        float3 Xj = make_float3(ptVbo[12*sphere2_id+0], ptVbo[12*sphere2_id+1], ptVbo[12*sphere2_id+2]);
        float ri = ptVbo[12*sphere1_id+9];
        float rj = ptVbo[12*sphere2_id+9];
        float3 toJ = Xj - Xi;
        float dist2 = toJ.x * toJ.x + toJ.y * toJ.y + toJ.z * toJ.z;
        if (dist2 < (rj + ri) * (rj + ri)) {
            respondSphereCollisions(ptVbo, f_tmp, sphere1_id, sphere2_id);
        }
    }
}

// host sided interface code
extern "C" 
{
    void launchParticleKernel(float *ptVbo, float *f_tmp,
                              float* sas_key1, float *sas_value1,
                              int numParticles, float t, float dT)
	{
        const int N_THREADS = 256;


        reset_force_buffer<<<ceil(numParticles/(float)N_THREADS), N_THREADS>>>(f_tmp, numParticles);

#ifdef SAS
        // bitonic sort
        // algorithm and colors refer to http://en.wikipedia.org/wiki/File:BitonicSort.svg
        SAS_init<<<ceil(numParticles/(float)N_THREADS), N_THREADS>>>(ptVbo, sas_key1, sas_value1, numParticles);
        unsigned int N = 2*numParticles;
        for (unsigned int B = 2; B <= N; B *= 2) {
            SAS_sort_brown<<<ceil(numParticles/(float)N_THREADS), N_THREADS>>>(sas_key1, sas_value1, N, B);
            for (unsigned int R = B/2; R >= 2; R /= 2) {
                SAS_sort_red<<<ceil(numParticles/(float)N_THREADS), N_THREADS>>>(sas_key1, sas_value1, N, R);
            }
        }
        // sort and sweep collision detection
        SAS_sphere_collisions<<<ceil(2*numParticles/(float)N_THREADS), N_THREADS>>>(ptVbo, f_tmp, sas_key1, sas_value1, numParticles);

#else
        // brute force collision detection
        sphere_collisions<<<ceil(numParticles/(float)N_THREADS), N_THREADS>>>(ptVbo, f_tmp, numParticles);
#endif

        box_collisions<<<ceil(numParticles/(float)N_THREADS), N_THREADS>>>(ptVbo, f_tmp, numParticles);
        integrate<<<ceil(numParticles/(float)N_THREADS), N_THREADS>>>(ptVbo, f_tmp, numParticles, dT);
    }
}
